#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/gpu_floyd.cuh"

__global__ void floyd_kernel(float* D, unsigned int V, unsigned int k) {
  if (blockDim.x * blockIdx.x + threadIdx.x < V && blockDim.y * blockIdx.y + threadIdx.y < V &&
      D[get_index((blockDim.x * blockIdx.x + threadIdx.x), (blockDim.y * blockIdx.y + threadIdx.y), V)] >
      D[get_index((blockDim.x * blockIdx.x + threadIdx.x), k, V)] +
      D[get_index(k, (blockDim.y * blockIdx.y + threadIdx.y), V)]) {
    D[get_index((blockDim.x * blockIdx.x + threadIdx.x), (blockDim.y * blockIdx.y + threadIdx.y), V)] =
        D[get_index((blockDim.x * blockIdx.x + threadIdx.x), k, V)] +
        D[get_index(k, (blockDim.y * blockIdx.y + threadIdx.y), V)];
  }
}

__global__ void floyd_shared_kernel(float* D, unsigned int V, unsigned int k) {
  // smem load
  __shared__ float k_x[n_thread];
  __shared__ float y_k[n_thread];
  if (threadIdx.y == 0) {
    if (blockDim.x * blockIdx.x + threadIdx.x < V) {
      k_x[threadIdx.x] = D[get_index((blockDim.x * blockIdx.x + threadIdx.x), k, V)];
    }
  }
  if (threadIdx.y == 1) {
    if (blockDim.y * blockIdx.y + threadIdx.x < V) {
      y_k[threadIdx.x] = D[get_index(k, (blockDim.y * blockIdx.y + threadIdx.x), V)];
    }
  }

  __syncthreads();
  if (blockDim.x * blockIdx.x + threadIdx.x < V && blockDim.y * blockIdx.y + threadIdx.y < V &&
      D[get_index((blockDim.x * blockIdx.x + threadIdx.x), (blockDim.y * blockIdx.y + threadIdx.y), V)] >
      k_x[threadIdx.x] + y_k[threadIdx.y]) {
    D[get_index((blockDim.x * blockIdx.x + threadIdx.x), (blockDim.y * blockIdx.y + threadIdx.y), V)] =
        k_x[threadIdx.x] + y_k[threadIdx.y];
  }
}
/*
__global__ void floyd_shared_kernel(float* D, unsigned int V, unsigned int k) {
  // smem load
  __shared__ float k_x[n_thread * n_unroll];
  __shared__ float y_k[n_thread * n_unroll];
  unsigned int tid = blockDim.x * threadIdx.y + threadIdx.x;
  if (threadIdx.y < (n_thread / 2)) {
    if (blockIdx.x * n_thread * n_unroll + tid < V)
      k_x[tid] = D[get_index((blockIdx.x * n_thread * n_unroll + tid), k, V)];
  } else {
    tid -= n_thread * n_unroll;
    if (blockIdx.y * n_thread * n_unroll + tid < V)
      y_k[tid] = D[get_index(k, (blockIdx.y * n_thread * n_unroll + tid), V)];
  }

  __syncthreads();
#pragma unroll
  for (unsigned int y = 0; y < n_unroll; y++) {
#pragma unroll
    for (unsigned int x = 0; x < n_unroll; x++) {
      if (blockDim.x * blockIdx.x * n_unroll + x * n_thread + threadIdx.x < V &&
          blockDim.y * blockIdx.y * n_unroll + y * n_thread + threadIdx.y < V &&
          D[get_index(
              (blockDim.x * blockIdx.x * n_unroll + x * n_thread + threadIdx.x),
              (blockDim.y * blockIdx.y * n_unroll + y * n_thread + threadIdx.y),
              V)] > k_x[threadIdx.x + n_thread * x] +
                        y_k[threadIdx.y + n_thread * y]) {
        D[get_index(
            (blockDim.x * blockIdx.x * n_unroll + x * n_thread + threadIdx.x),
            (blockDim.y * blockIdx.y * n_unroll + y * n_thread + threadIdx.y),
            V)] =
            k_x[threadIdx.x + n_thread * x] + y_k[threadIdx.y + n_thread * y];
      }
    }
  }
}
*/
__global__ void init_kernel(float* D, unsigned int V, float inf) {
  D[get_index(
      (blockIdx.x * blockDim.x + threadIdx.x),
      (blockIdx.y * blockDim.y + threadIdx.y), V)] =
      inf;
}

__global__ void init_kernel_unroll(float* D, unsigned int V, float inf) {
#pragma unroll
  for (unsigned int y = 0; y < n_unroll; y++) {
#pragma unroll
    for (unsigned int x = 0; x < n_unroll; x++) {
      D[get_index(
          (blockIdx.x * blockDim.x * n_unroll + x * n_thread + threadIdx.x),
          (blockIdx.y * blockDim.y * n_unroll + y * n_thread + threadIdx.y), V)] =
          inf;
    }
  }
}

__global__ void const_kernel(unsigned int* src, unsigned int* dst, float* w,
                             float* D, unsigned int V) {
  D[get_index(dst[blockIdx.x * blockDim.x + threadIdx.x],
              src[blockIdx.x * blockDim.x + threadIdx.x], V)] =
      w[blockIdx.x * blockDim.x + threadIdx.x];
}

__global__ void zero_kernel(float* D, unsigned int V) {
  D[get_index((blockIdx.x * blockDim.x + threadIdx.x),
              (blockIdx.x * blockDim.x + threadIdx.x), V)] = 0.0;
}

__host__ float* gpu_floyd(unsigned int* src, unsigned int* dst, float* w,
                        unsigned int V, unsigned int E) {
  float* hmem = (float*)malloc(V * V * sizeof(float));
  float* D;
  unsigned int* d_src;
  unsigned int* d_dst;
  float* d_w;
  CHK(hipMalloc(&D, V * V * sizeof(float)), "hipMalloc");
  CHK(hipMalloc(&d_src, E * sizeof(unsigned int)), "cudaMalloc_src");
  CHK(hipMalloc(&d_dst, E * sizeof(unsigned int)), "cudaMalloc_dst");
  CHK(hipMalloc(&d_w, E * sizeof(float)), "cudaMalloc_w");
  CHK(hipMemset(D, 0, V * V * sizeof(float)), "hipMemset");
  CHK(hipMemcpy(d_src, src, E * sizeof(unsigned int), hipMemcpyHostToDevice),
      "cudaMemcpyH2D_src");
  CHK(hipMemcpy(d_dst, dst, E * sizeof(unsigned int), hipMemcpyHostToDevice),
      "cudaMemcpyH2D_dst");
  CHK(hipMemcpy(d_w, w, E * sizeof(float), hipMemcpyHostToDevice),
      "cudaMemcpyH2D_w");

  // configuration
  dim3 b_conf(n_thread, n_thread);
  unsigned int n_blocks = div_up(V, n_thread);
  dim3 g_conf(n_blocks, n_blocks);

  // initialize D (construct from src, dst, w) on GPU
  CHK((init_kernel<<<g_conf, b_conf>>>(D, V, INFINITY)), "initialization");
  CHK((const_kernel<<<div_up(E, 1024), 1024>>>(d_src, d_dst, d_w, D, V)),
      "construct");
  unsigned int v_block = V < 1024 ? V : 1024;
  CHK((zero_kernel<<<div_up(V, v_block), v_block>>>(D, V)), "zero");

  for (unsigned int i = 0; i < V; i++) {
    // one kernel for each interation of k
    CHK((floyd_kernel<<<g_conf, b_conf>>>(D, V, i)), "kernel");
  }
  CHK(hipMemcpy(hmem, D, V * V * sizeof(float), hipMemcpyDeviceToHost),
      "cudaMemcpyD2H");
  CHK(hipFree(D), "freeD");
  CHK(hipFree(d_src), "freesrc");
  CHK(hipFree(d_dst), "freedst");
  CHK(hipFree(d_w), "freedist");
  return hmem;
}

__host__ float* gpu_floyd_shared(unsigned int* src, unsigned int* dst, float* w,
                        unsigned int V, unsigned int E) {
  float* hmem = (float*)malloc(V * V * sizeof(float));
  float* D;
  unsigned int* d_src;
  unsigned int* d_dst;
  float* d_w;
  CHK(hipMalloc(&D, V * V * sizeof(float)), "hipMalloc");
  CHK(hipMalloc(&d_src, E * sizeof(unsigned int)), "cudaMalloc_src");
  CHK(hipMalloc(&d_dst, E * sizeof(unsigned int)), "cudaMalloc_dst");
  CHK(hipMalloc(&d_w, E * sizeof(float)), "cudaMalloc_w");
  CHK(hipMemset(D, 0, V * V * sizeof(float)), "hipMemset");
  CHK(hipMemcpy(d_src, src, E * sizeof(unsigned int), hipMemcpyHostToDevice),
      "cudaMemcpyH2D_src");
  CHK(hipMemcpy(d_dst, dst, E * sizeof(unsigned int), hipMemcpyHostToDevice),
      "cudaMemcpyH2D_dst");
  CHK(hipMemcpy(d_w, w, E * sizeof(float), hipMemcpyHostToDevice),
      "cudaMemcpyH2D_w");

  // configuration
  dim3 b_conf(n_thread, n_thread);
  // unsigned int n_blocks = div_up(V, (n_thread * n_unroll));
  unsigned int n_blocks = div_up(V, n_thread);
  dim3 g_conf(n_blocks, n_blocks);

  // initialize D (construct from src, dst, w) on GPU
  CHK((init_kernel<<<g_conf, b_conf>>>(D, V, INFINITY)), "initialization");
  CHK((const_kernel<<<div_up(E, 1024), 1024>>>(d_src, d_dst, d_w, D, V)),
      "construct");
  unsigned int v_block = V < 1024 ? V : 1024;
  CHK((zero_kernel<<<div_up(V, v_block), v_block>>>(D, V)), "zero");

  for (unsigned int i = 0; i < V; i++) {
    // one kernel for each interation of k
    CHK((floyd_shared_kernel<<<g_conf, b_conf>>>(D, V, i)), "kernel");
  }
  CHK(hipMemcpy(hmem, D, V * V * sizeof(float), hipMemcpyDeviceToHost),
      "cudaMemcpyD2H");
  CHK(hipFree(D), "freeD");
  CHK(hipFree(d_src), "freesrc");
  CHK(hipFree(d_dst), "freedst");
  CHK(hipFree(d_w), "freedist");
  return hmem;
}
