#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/gpu_floyd.cuh"

__global__ void floyd_kernel(float* D, unsigned int V, unsigned int k) {
  // smem load
  __shared__ float x_k[n_thread * n_unroll];
  __shared__ float k_y[n_thread * n_unroll];
  unsigned int tid = blockDim.x * threadIdx.y + threadIdx.x;
  if (threadIdx.y < (n_thread / 2)) {
    if (blockIdx.x * n_thread * n_unroll + tid < V)
      x_k[tid] = D[get_index(blockIdx.x * n_thread * n_unroll + tid, k, V)];
  } else {
    tid -= n_thread * n_unroll;
    if (blockIdx.y * n_thread * n_unroll + tid < V)
      k_y[tid] = D[get_index(k, blockIdx.y * n_thread * n_unroll + tid, V)];
  }

  __syncthreads();
#pragma unroll
  for (unsigned int y = 0; y < n_unroll; y++) {
#pragma unroll
    for (unsigned int x = 0; x < n_unroll; x++) {
      if (blockDim.x * blockIdx.x * n_unroll + x * n_thread + threadIdx.x < V &&
          blockDim.y * blockIdx.y * n_unroll + y * n_thread + threadIdx.y < V &&
          D[get_index(
              blockDim.x * blockIdx.x * n_unroll + x * n_thread + threadIdx.x,
              blockDim.y * blockIdx.y * n_unroll + y * n_thread + threadIdx.y,
              V)] > x_k[threadIdx.x + n_thread * x] +
                        k_y[threadIdx.y + n_thread * y]) {
        D[get_index(
            blockDim.x * blockIdx.x * n_unroll + x * n_thread + threadIdx.x,
            blockDim.y * blockIdx.y * n_unroll + y * n_thread + threadIdx.y,
            V)] =
            x_k[threadIdx.x + n_thread * x] + k_y[threadIdx.y + n_thread * y];
      }
    }
  }
}

__global__ void init_kernel(float* D, unsigned int V, float inf) {
#pragma unroll
  for (unsigned int y = 0; y < n_unroll; y++) {
#pragma unroll
    for (unsigned int x = 0; x < n_unroll; x++) {
      D[get_index(
          blockIdx.x * blockDim.x * n_unroll + x * n_thread + threadIdx.x,
          blockIdx.y * blockDim.y * n_unroll + y * n_thread + threadIdx.y, V)] =
          inf;
    }
  }
}

__global__ void const_kernel(unsigned int* src, unsigned int* dst, float* w,
                             float* D, unsigned int V) {
  D[get_index(src[blockIdx.x * blockDim.x + threadIdx.x],
              dst[blockIdx.x * blockDim.x + threadIdx.x], V)] =
      w[blockIdx.x * blockDim.x + threadIdx.x];
}

__host__ float* gpu_floyd(unsigned int* src, unsigned int* dst, float* w,
                        unsigned int V, unsigned int E) {
  float* hmem = (float*)malloc(V * V * sizeof(float));
  float* D;
  unsigned int* d_src;
  unsigned int* d_dst;
  float* d_w;
  CHK(hipMalloc(&D, V * V * sizeof(float)), "hipMalloc");
  CHK(hipMalloc(&d_src, E * sizeof(unsigned int)), "cudaMalloc_src");
  CHK(hipMalloc(&d_dst, E * sizeof(unsigned int)), "cudaMalloc_dst");
  CHK(hipMalloc(&d_w, E * sizeof(float)), "cudaMalloc_w");
  CHK(hipMemset(D, 0, V * V * sizeof(float)), "hipMemset");
  CHK(hipMemcpy(d_src, src, E * sizeof(unsigned int), hipMemcpyHostToDevice),
      "cudaMemcpyH2D_src");
  CHK(hipMemcpy(d_dst, dst, E * sizeof(unsigned int), hipMemcpyHostToDevice),
      "cudaMemcpyH2D_dst");
  CHK(hipMemcpy(d_w, w, E * sizeof(float), hipMemcpyHostToDevice),
      "cudaMemcpyH2D_w");

  // configuration
  dim3 b_conf(n_thread, n_thread);
  unsigned int n_blocks = div_up(V, (n_thread * n_unroll));
  dim3 g_conf(n_blocks, n_blocks);

  // initialize D (construct from src, dst, w) on GPU
  CHK((init_kernel<<<g_conf, b_conf>>>(D, V, INF)), "initialization");
  CHK((const_kernel<<<div_up(E, 1024), 1024>>>(d_src, d_dst, d_w, D, V)),
      "construct");

  for (unsigned int i = 0; i < V; i++) {
    // one kernel for each interation of k
    CHK((floyd_kernel<<<g_conf, b_conf>>>(D, V, i)), "kernel");
  }
  CHK(hipMemcpy(hmem, D, V * V * sizeof(float), hipMemcpyDeviceToHost),
      "cudaMemcpyD2H");
  return hmem;
}
